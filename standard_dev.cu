#include "hip/hip_runtime.h"

#include <hip/hip_runtime.h>
#include <>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <ctime>
#include <math.h>

#define M 1024//threads per block

__global__ void blockSummer(int* mainVec, int* blockSum, int N, int mean, bool mode) {
	__shared__ int sumVec[M + 1];
	int tid = threadIdx.x;
	int eid = blockIdx.x * blockDim.x + threadIdx.x;
	if (eid < N) {
		if (mode == true) {
			sumVec[tid] = (mainVec[eid] - mean) * (mainVec[eid] - mean);
		}
		else {
			sumVec[tid] = mainVec[eid];
		}
	}
	else {
		sumVec[tid] = 0;
	}
	__syncthreads();
	for (unsigned int i = 1; i < blockDim.x; i *= 2) {
		if (tid % (2 * i) == 0) {
			sumVec[tid] += sumVec[tid + i];
		}
		__syncthreads();
	}
	if (tid == 0) {
		blockSum[blockIdx.x] = sumVec[0];
	}
}


void randomInts(int* vector, int length) {
	for (size_t i = 0; i < length; i++) {
		vector[i] = rand() % 100;
	}
}

int main() {
	clock_t start, stop;
	int N = 95000000;
	int numOfBlocks = (N + M - 1) / M;
	int* mainVec = (int*)malloc(N * sizeof(int));
	int* intermediate = (int*)malloc(N * sizeof(int));
	int* blockSum = (int*)malloc(numOfBlocks * sizeof(int));
	int* blockSum2 = (int*)malloc(numOfBlocks * sizeof(int));
	randomInts(mainVec, N);
	int* d_mainVec, * d_blockSum, * d_blockSum2;
	hipMalloc(&d_mainVec, N * sizeof(int));
	hipMalloc(&d_blockSum, numOfBlocks * sizeof(int));

	//printf("\n*************PARALLEL EXECUTION*************/n");

	hipMemcpy(d_mainVec, mainVec, N * sizeof(int), hipMemcpyHostToDevice);
	start = std::clock();
	blockSummer << <numOfBlocks, M >> > (d_mainVec, d_blockSum, N, 0, false);
	hipDeviceSynchronize();
	hipMemcpy(blockSum, d_blockSum, numOfBlocks * sizeof(int), hipMemcpyDeviceToHost);
	int sum = 0;
	for (size_t i = 0; i < numOfBlocks; i++) {
		sum += blockSum[i];
	}
	int mean = (int)(sum / N);
	blockSummer << <numOfBlocks, M >> > (d_mainVec, d_blockSum, N, mean, true);
	hipDeviceSynchronize();
	hipMemcpy(blockSum, d_blockSum, numOfBlocks * sizeof(int), hipMemcpyDeviceToHost);
	int stddevsum = 0;
	for (size_t i = 0; i < numOfBlocks; i++) {
		stddevsum += blockSum[i];
	}
	float stddev = (stddevsum / N);
	float finalstddev = sqrt(stddev);
	printf("\nStandard Deviation : %f", finalstddev);
	
	
	free(mainVec);
	free(blockSum);
	free(intermediate);
	hipFree(d_mainVec);
	hipFree(d_blockSum);

	return 0;
}
