
#include<stdio.h>
#include <hip/hip_runtime.h>

void random_ints(int* a, int N) 
{
	int i;
	for (i = 0; i < N; ++i)
		a[i] = rand()%20000;
}

void add_array(int* a, int N)  
{
	for (int i = 0; i < N; ++i)
		a[i] =i;
}
__global__ void binary_search(int* a, int* b, bool* c, int sizeofa) //kernal function
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	printf(" %d\n", index);
	int key = b[index];
	int min = 0, max = sizeofa;
	int mid = sizeofa / 2;
	while (min != mid)
	{
		if (key == a[mid])
		{
			break;
		}
		else if (key < a[mid])
		{
			min = min;
			max = mid;
		}
		else {
			min = mid;
			max = max;
		}
		mid = (min + max) / 2;
	}
	
	if (key == a[mid])
		c[index] = true;
	else
		c[index] = false;

	printf(" %d %d  %d %d\n", index, key, a[mid],c[index]);
}

int main()
{
	int N = 10000; //size of given array
	int M = 1000; //Number of searching element
	size_t size = N * sizeof(int);
	size_t size2 = M * sizeof(int);

	//allocate memory for host array
	int* vector1 = (int*)malloc(size);
	int* vector2 = (int*)malloc(size2);
	bool* vector3 = (bool*)malloc(M * sizeof(bool));

	//insert number into array
	add_array(vector1,N);

	//insert random elements to search
	random_ints(vector2,M);

	//create device array pointer
	int *d_vector1;
	int *d_vector2;
	bool *d_vector3;

	//allocate device memory for vector
	hipMalloc(& d_vector1, size);
	hipMalloc(& d_vector2, size2);
	hipMalloc(& d_vector3, M*sizeof(bool));

	//copy vectors from host memory to dvice memory
	hipMemcpy(d_vector1,vector1, size, hipMemcpyHostToDevice);
	hipMemcpy(d_vector2,vector2, size2, hipMemcpyHostToDevice);

	//call kernal
	binary_search<<<M,1>>>(d_vector1, d_vector2, d_vector3,N);
	

	hipMemcpy(vector3,d_vector3, M * sizeof(bool), hipMemcpyDeviceToHost);

	for (int i = 0; i < M; i++)
	{
		if(vector3[i]==true)
			printf("%d is present in array\n",vector2[i]);
		else if(vector3[i] == 0)
			printf("%d is not present in array\n", vector2[i]);

	}
}
