#include<stdio.h>
#include <hip/hip_runtime.h>

void random_ints(int* a, int N)
{
	int i;
	for (i = 0; i < N; ++i)
		a[i] = rand()%100;
}

__global__ void add_vector(int* a,int* b,int*c)
{
	int i = blockIdx.x*blockDim.x+ threadIdx.x;
	c[i] = a[i] + b[i];
}

int main()
{
	int N = 10000; //size of vector
	int M = 10; //Number of thread
	size_t size = N * sizeof(int);
	//allocate memory for host vector
	int* vector1 = (int*)malloc(size);
	int* vector2 = (int*)malloc(size);
	int* vector3 = (int*)malloc(size);

	//insert number into vector
	random_ints(vector1,N);
	random_ints(vector2,N);

	//create device vector pointer
	int *d_vector1;
	int *d_vector2;
	int *d_vector3;

	//allocate device memory for vector
	hipMalloc(& d_vector1, size);
	hipMalloc(& d_vector2, size);
	hipMalloc(& d_vector3, size);

	//copy vectors from host memory to dvice memory
	hipMemcpy(d_vector1,vector1, size, hipMemcpyHostToDevice);
	hipMemcpy(d_vector2,vector2, size, hipMemcpyHostToDevice);

	//call kernal
	add_vector<<<(N+M-1)/M,M>>>(d_vector1, d_vector2, d_vector3);
	
	//cudaDeviceSynchronize();
	hipMemcpy(vector3,d_vector3, size, hipMemcpyDeviceToHost);

	for (int i = 0; i < 10000; i++)
	{
		printf("%d %d + %d =%d\n",i,vector1[i], vector2[i], vector3[i]);

	}
}
